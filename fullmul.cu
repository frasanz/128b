#include "hip/hip_runtime.h"
/* It sums 5 times two 128bits integers using CUDA */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "show.h"

__global__ void mul_uint128(my_uint128_t * op1, my_uint128_t * op2, my_uint128_t * result)
{
 asm ("{\n\t"
         "mul.lo.u32      %0, %4, %8;    \n\t"
         "mul.hi.u32      %1, %4, %8;    \n\t"
         "mad.lo.cc.u32   %1, %4, %9, %1;\n\t"
         "madc.hi.u32     %2, %4, %9,  0;\n\t"
         "mad.lo.cc.u32   %1, %5, %8, %1;\n\t"
         "madc.hi.cc.u32  %2, %5, %8, %2;\n\t"
         "madc.hi.u32     %3, %4,%10,  0;\n\t"
         "mad.lo.cc.u32   %2, %4,%10, %2;\n\t"
         "madc.hi.u32     %3, %5, %9, %3;\n\t"
         "mad.lo.cc.u32   %2, %5, %9, %2;\n\t"
         "madc.hi.u32     %3, %6, %8, %3;\n\t"
         "mad.lo.cc.u32   %2, %6, %8, %2;\n\t"
         "madc.lo.u32     %3, %4,%11, %3;\n\t"
         "mad.lo.u32      %3, %5,%10, %3;\n\t"
         "mad.lo.u32      %3, %6, %9, %3;\n\t"
         "mad.lo.u32      %3, %7, %8, %3;\n\t"
         "}"
         : "=r"(result->x), "=r"(result->y), "=r"(result->z), "=r"(result->w)
         : "r"(op1->x), "r"(op1->y), "r"(op1->z), "r"(op1->w),
           "r"(op2->x), "r"(op2->y), "r"(op2->z), "r"(op2->w));


	printf("%u %u %u %u %d\n",result->w, result->z, result->y, result->x, threadIdx.x);
}


int main()
{

	/* Definition of operands and result */
	my_uint128_t * d_op1;
	my_uint128_t * h_op1;
	my_uint128_t * d_op2;
	my_uint128_t * h_op2;
	my_uint128_t * d_result;
	my_uint128_t * h_result;

	/* Malloc */
	hipMalloc((void**)&d_op1, sizeof(my_uint128_t));
	h_op1 = (my_uint128_t *) malloc(sizeof(my_uint128_t));


	hipMalloc((void**)&d_op2, sizeof(my_uint128_t));
	h_op2 = (my_uint128_t *) malloc(sizeof(my_uint128_t));


	hipMalloc((void**)&d_result, sizeof(my_uint128_t));
	h_result = (my_uint128_t *) malloc(sizeof(my_uint128_t));

	/* Definition of op1 and op2 */
	h_op1->x = 0;
	h_op1->y = 1001203;
	h_op1->z = 0;
	h_op1->w = 0;
	h_op2->x = 0;
	h_op2->y = 0;
	h_op2->z = 3211;
	h_op2->w = 0;
	hipMemcpy(d_op1, h_op1, sizeof(my_uint128_t), hipMemcpyHostToDevice);
	hipMemcpy(d_op2, h_op2, sizeof(my_uint128_t), hipMemcpyHostToDevice);

	mul_uint128<<<1,5>>>(d_op1, d_op2, d_result);
	hipMemcpy(h_result, d_result, sizeof(my_uint128_t), hipMemcpyDeviceToHost);

	printf("%u %u %u %u \n",h_result->w, h_result->z, h_result->y, h_result->x);
	show(h_op1);
	show(h_op2);
	show(h_result);
	hipFree(d_result);
	hipFree(d_op1);
	hipFree(d_op2);
	free(h_result);
	free(h_op1);
	free(h_op2);
	return 0;
}
