/* It sums 5 times two 128bits integers using CUDA */

#include <stdio.h>
#include <hip/hip_runtime.h>

typedef uint4 my_uint128_t;
__global__ void add_uint128(float f)
{
	my_uint128_t addend;
	my_uint128_t augend;
	my_uint128_t res;

	/* Posicion numero 
	.w .z .y .x*/

	addend.x = 0;
	addend.y = 1001203;
	addend.z = 0;
	addend.w = 0;
	augend.x = 0;
	augend.y = 0;
	augend.z = 3211;
	augend.w = 0;
	res.x = 0;
	res.y = 0;
	res.z = 0;
	res.w = 0;

 asm ("{\n\t"
         "mul.lo.u32      %0, %4, %8;    \n\t"
         "mul.hi.u32      %1, %4, %8;    \n\t"
         "mad.lo.cc.u32   %1, %4, %9, %1;\n\t"
         "madc.hi.u32     %2, %4, %9,  0;\n\t"
         "mad.lo.cc.u32   %1, %5, %8, %1;\n\t"
         "madc.hi.cc.u32  %2, %5, %8, %2;\n\t"
         "madc.hi.u32     %3, %4,%10,  0;\n\t"
         "mad.lo.cc.u32   %2, %4,%10, %2;\n\t"
         "madc.hi.u32     %3, %5, %9, %3;\n\t"
         "mad.lo.cc.u32   %2, %5, %9, %2;\n\t"
         "madc.hi.u32     %3, %6, %8, %3;\n\t"
         "mad.lo.cc.u32   %2, %6, %8, %2;\n\t"
         "madc.lo.u32     %3, %4,%11, %3;\n\t"
         "mad.lo.u32      %3, %5,%10, %3;\n\t"
         "mad.lo.u32      %3, %6, %9, %3;\n\t"
         "mad.lo.u32      %3, %7, %8, %3;\n\t"
         "}"
         : "=r"(res.x), "=r"(res.y), "=r"(res.z), "=r"(res.w)
         : "r"(addend.x), "r"(addend.y), "r"(addend.z), "r"(addend.w),
           "r"(augend.x), "r"(augend.y), "r"(augend.z), "r"(augend.w));


	printf("%u %u %u %u %d\n",res.w, res.z, res.y, res.x, threadIdx.x);
}


int main()
{
	add_uint128<<<1,5>>>(1.1f);
	hipDeviceReset();
	return 0;
}
