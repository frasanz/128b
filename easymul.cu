#include "hip/hip_runtime.h"
/* It sums 5 times two 128bits integers using CUDA */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "show.h"

__global__ void mul_uint128(my_uint128_t * result)
{
	my_uint128_t a,b;

	/* Posicion numero 
	.w .z .y .x*/

	a.x = 0;
	a.y = 1001203;
	a.z = 0;
	a.w = 0;
	b.x = 0;
	b.y = 0;
	b.z = 3211;
	b.w = 0;

 asm ("{\n\t"
         "mul.lo.u32      %0, %4, %8;    \n\t"
         "mul.hi.u32      %1, %4, %8;    \n\t"
         "mad.lo.cc.u32   %1, %4, %9, %1;\n\t"
         "madc.hi.u32     %2, %4, %9,  0;\n\t"
         "mad.lo.cc.u32   %1, %5, %8, %1;\n\t"
         "madc.hi.cc.u32  %2, %5, %8, %2;\n\t"
         "madc.hi.u32     %3, %4,%10,  0;\n\t"
         "mad.lo.cc.u32   %2, %4,%10, %2;\n\t"
         "madc.hi.u32     %3, %5, %9, %3;\n\t"
         "mad.lo.cc.u32   %2, %5, %9, %2;\n\t"
         "madc.hi.u32     %3, %6, %8, %3;\n\t"
         "mad.lo.cc.u32   %2, %6, %8, %2;\n\t"
         "madc.lo.u32     %3, %4,%11, %3;\n\t"
         "mad.lo.u32      %3, %5,%10, %3;\n\t"
         "mad.lo.u32      %3, %6, %9, %3;\n\t"
         "mad.lo.u32      %3, %7, %8, %3;\n\t"
         "}"
         : "=r"(result->x), "=r"(result->y), "=r"(result->z), "=r"(result->w)
         : "r"(a.x), "r"(a.y), "r"(a.z), "r"(a.w),
           "r"(b.x), "r"(b.y), "r"(b.z), "r"(b.w));


	printf("%u %u %u %u %d\n",result->w, result->z, result->y, result->x, threadIdx.x);
}


int main()
{
	my_uint128_t * d_result;
	my_uint128_t * h_result;
	hipMalloc((void**)&d_result, sizeof(my_uint128_t));
	h_result = (my_uint128_t *) malloc(sizeof(my_uint128_t));

	mul_uint128<<<1,5>>>(d_result);
	hipMemcpy(h_result, d_result, sizeof(my_uint128_t), hipMemcpyDeviceToHost);

	printf("%u %u %u %u \n",h_result->w, h_result->z, h_result->y, h_result->x);
	show(h_result);
	hipFree(d_result);
	free(h_result);
	return 0;
}
