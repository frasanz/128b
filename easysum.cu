#include "hip/hip_runtime.h"
/* It sums 5 times two 128bits integers using CUDA */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "show.h"

__global__ void add_uint128(my_uint128_t * result)
{
	my_uint128_t a, b;

	/* Posicion numero 
	.w .z .y .x*/


	a.x = 0;
	a.y = 0;
	a.z = 4294967295;
	a.w = 0;
	b.x = 0;
	b.y = 0;
	b.z = 1;
	b.w = 0;

	asm ("add.cc.u32      %0, %4, %8;\n\t"
			"addc.cc.u32     %1, %5, %9;\n\t"
			"addc.cc.u32     %2, %6, %10;\n\t"
			"addc.u32        %3, %7, %11;\n\t"
			: "=r"(result->x), "=r"(result->y), "=r"(result->z), "=r"(result->w)
			: "r"(a.x), "r"(a.y), "r"(a.z), "r"(a.w),
			"r"(b.x), "r"(b.y), "r"(b.z), "r"(b.w)); 

	printf("%u %u %u %u %d\n",result->w, result->z, result->y, result->x, threadIdx.x);
}


int main()
{
	my_uint128_t * result;
	my_uint128_t * h_result;
	hipMalloc((void**)&result, sizeof(my_uint128_t));
	h_result = (my_uint128_t *) malloc(sizeof(my_uint128_t));
	add_uint128<<<1,5>>>(result);
	hipMemcpy(h_result, result, sizeof(my_uint128_t), hipMemcpyDeviceToHost);
	printf("%u %u %u %u \n",h_result->w, h_result->z, h_result->y, h_result->x);
	show(h_result);
	hipFree(result);
	free(h_result);

	return 0;
}
