#include <stdio.h>
#include <hip/hip_runtime.h>

typedef uint4 my_uint128_t;
__global__ void add_uint128(float f)
{
	my_uint128_t addend;
	my_uint128_t augend;
	my_uint128_t res;

	/* Posicion numero 
	.w .z .y .x*/


	addend.x = 0;
	addend.y = 0;
	addend.z = 1;
	addend.w = 0;
	augend.x = 0;
	augend.y = 0;
	augend.z = 4294967294;
	augend.w = 0;
	res.x = 0;
	res.y = 0;
	res.z = 0;
	res.w = 0;

	asm ("add.cc.u32      %0, %4, %8;\n\t"
			"addc.cc.u32     %1, %5, %9;\n\t"
			"addc.cc.u32     %2, %6, %10;\n\t"
			"addc.u32        %3, %7, %11;\n\t"
			: "=r"(res.x), "=r"(res.y), "=r"(res.z), "=r"(res.w)
			: "r"(addend.x), "r"(addend.y), "r"(addend.z), "r"(addend.w),
			"r"(augend.x), "r"(augend.y), "r"(augend.z), "r"(augend.w)); 

	printf("%d %d %d %d %d\n",res.x, res.y, res.z, res.w, threadIdx.x);
	printf("%8x %8x %8x %8x %d\n",res.w, res.z, res.y, res.x, threadIdx.x);
}


int main()
{
	add_uint128<<<1,5>>>(1.1f);
	hipDeviceReset();
	return 0;
}
