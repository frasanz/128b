/* It sums 5 times two 128bits integers using CUDA */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <gmp.h>

typedef uint4 my_uint128_t;
__global__ void add_uint128(my_uint128_t * result)
{
	my_uint128_t a, b;

	/* Posicion numero 
	.w .z .y .x*/


	a.x = 0;
	a.y = 0;
	a.z = 4294967295;
	a.w = 0;
	b.x = 0;
	b.y = 0;
	b.z = 1;
	b.w = 0;

	asm ("add.cc.u32      %0, %4, %8;\n\t"
			"addc.cc.u32     %1, %5, %9;\n\t"
			"addc.cc.u32     %2, %6, %10;\n\t"
			"addc.u32        %3, %7, %11;\n\t"
			: "=r"(result->x), "=r"(result->y), "=r"(result->z), "=r"(result->w)
			: "r"(a.x), "r"(a.y), "r"(a.z), "r"(a.w),
			"r"(b.x), "r"(b.y), "r"(b.z), "r"(b.w)); 

	printf("%u %u %u %u %d\n",result->w, result->z, result->y, result->x, threadIdx.x);
}


int main()
{
	mpz_t fullresult;
	mpz_init(fullresult);
	mpz_t  dos_32, dos_64, dos_96;
	mpz_init_set_str (dos_32, "4294967296", 10);
	mpz_init_set_str (dos_64, "18446744073709551616", 10);
	mpz_init_set_str (dos_96, "79228162514264337593543950336", 10);
	my_uint128_t * result;
	my_uint128_t * h_result;
	hipMalloc((void**)&result, sizeof(my_uint128_t));
	h_result = (my_uint128_t *) malloc(sizeof(my_uint128_t));
	h_result->x=10;
	h_result->y=11;
	h_result->z=12;
	h_result->w=13;
	add_uint128<<<1,5>>>(result);
	hipMemcpy(h_result, result, sizeof(my_uint128_t), hipMemcpyDeviceToHost);
	printf("%u %u %u %u \n",h_result->w, h_result->z, h_result->y, h_result->x);
	mpz_add_ui(fullresult, fullresult, h_result->x);
	mpz_addmul_ui(fullresult, dos_32, h_result->y);
	mpz_addmul_ui(fullresult, dos_64, h_result->z);
	mpz_addmul_ui(fullresult, dos_96, h_result->w); 
	gmp_printf ("fullresult %Zd\n",fullresult );
	hipFree(result);
	free(h_result);

	return 0;
}
