#include <stdio.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "show.h"

int show(my_uint128_t * value){
	mpz_t fullresult;
	mpz_init(fullresult);
	mpz_t  dos_32, dos_64, dos_96;
	mpz_init_set_str (dos_32, "4294967296", 10);
	mpz_init_set_str (dos_64, "18446744073709551616", 10);
	mpz_init_set_str (dos_96, "79228162514264337593543950336", 10);
	mpz_add_ui(fullresult, fullresult, value->x);
	mpz_addmul_ui(fullresult, dos_32, value->y);
	mpz_addmul_ui(fullresult, dos_64, value->z);
	mpz_addmul_ui(fullresult, dos_96, value->w); 
	gmp_printf ("fullresult %Zd\n",fullresult );

	return 0;
}
